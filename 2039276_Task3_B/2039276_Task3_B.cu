#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

/*
compile with c++ lodepng file:
nvcc 2039276_Task3_B.cu lodepng.cpp -o 2039276_Task3_B

to run:
./2039276_Task3_B

*/



__device__ unsigned int d_width;

__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4);
	return image[i];
  }
  
  __device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4) +1;
	return image[i];
  }
  
  __device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4) +2;
	return image[i];
  }
  
  __device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col){
	unsigned int i = (row * d_width * 4) + (col * 4) +3;
	return image[i];
  }
  
  __device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red){
	unsigned int i = (row * d_width * 4) + (col * 4);
	image[i] = red;
  }
  
  __device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green){
	unsigned int i = (row * d_width * 4) + (col * 4) +1;
	image[i] = green;
  }
  
  __device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue){
	unsigned int i = (row * d_width * 4) + (col * 4) +2;
	image[i] = blue;
  }
  
  __device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha){
	unsigned int i = (row * d_width * 4) + (col * 4) +3;
	image[i] = alpha;
  }
  
__global__ void square(unsigned char * gpu_imageOutput, unsigned char * gpu_imageInput, unsigned int *width){
	
	unsigned redTL, redTC, redTR;
	unsigned redL, redC, redR;
	unsigned redBL, redBC, redBR;
	unsigned newRed;

	unsigned greenTL, greenTC, greenTR;
	unsigned greenL, greenC, greenR;
	unsigned greenBL, greenBC, greenBR;
	unsigned newGreen;

	unsigned blueTL, blueTC, blueTR;
	unsigned blueL, blueC, blueR;
	unsigned blueBL, blueBC, blueBR;
	unsigned newBlue;

	int row = blockIdx.x+1;
	int col = threadIdx.x+1;

	d_width = *width;
	
	setGreen(gpu_imageOutput, row, col, getGreen(gpu_imageInput, row, col));
	setBlue(gpu_imageOutput, row, col, getBlue(gpu_imageInput, row, col));
	setAlpha(gpu_imageOutput, row, col, 255);

	redTL = getRed(gpu_imageInput, row - 1, col - 1);
	redTC = getRed(gpu_imageInput, row - 1, col);
	redTR = getRed(gpu_imageInput, row - 1, col + 1);

	redL = getRed(gpu_imageInput, row, col - 1);
	redC = getRed(gpu_imageInput, row, col);
	redR = getRed(gpu_imageInput, row, col + 1);

	redBL = getRed(gpu_imageInput, row + 1, col - 1);
	redBC = getRed(gpu_imageInput, row + 1, col);
	redBR = getRed(gpu_imageInput, row + 1, col + 1);
	
	//Bluring red color value
	newRed = (redTL+redTC+redTR+redL+redC+redR+redBL+redBC+redBR)/9;  

	setRed(gpu_imageOutput, row, col, newRed);

	greenTL = getGreen(gpu_imageInput, row - 1, col - 1);
	greenTC = getGreen(gpu_imageInput, row - 1, col);
	greenTR = getGreen(gpu_imageInput, row - 1, col + 1);

	greenL = getGreen(gpu_imageInput, row, col - 1);
	greenC = getGreen(gpu_imageInput, row, col);
	greenR = getGreen(gpu_imageInput, row, col + 1);

	greenBL = getGreen(gpu_imageInput, row + 1, col - 1);
	greenBC = getGreen(gpu_imageInput, row + 1, col);
	greenBR = getGreen(gpu_imageInput, row + 1, col + 1);

	//Bluring green color value
	newGreen = (greenTL+greenTC+greenTR+greenL+greenC+greenR+greenBL+greenBC+greenBR)/9; 

	setGreen(gpu_imageOutput, row, col, newGreen);

	blueTL = getBlue(gpu_imageInput, row - 1, col - 1);
	blueTC = getBlue(gpu_imageInput, row - 1, col);
	blueTR = getBlue(gpu_imageInput, row - 1, col + 1);

	blueL = getBlue(gpu_imageInput, row, col - 1);
	blueC = getBlue(gpu_imageInput, row, col);
	blueR = getBlue(gpu_imageInput, row, col + 1);

	blueBL = getBlue(gpu_imageInput, row + 1, col - 1);
	blueBC = getBlue(gpu_imageInput, row + 1, col);
	blueBR = getBlue(gpu_imageInput, row + 1, col + 1);

	//Bluring blue color value
	newBlue = (blueTL+blueTC+blueTR+blueL+blueC+blueR+blueBL+blueBC+blueBR)/9; 

	setBlue(gpu_imageOutput, row, col, newBlue);
}

int main(int argc, char **argv){

	unsigned char *image;
	unsigned int width;
	unsigned int height;
	const char* filename = "hck.png";
	const char* newFileName = "filtered.png";

	//Decoding Image
	lodepng_decode32_file(&image, &width, &height, filename);

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	//Declaring gpuImageWidth and setting the value 
	unsigned int* d_wid; 
	hipMalloc( (void**) &d_wid, sizeof(int));
	hipMemcpy(d_wid, &width, sizeof(int), hipMemcpyHostToDevice);

	// launch the kernel
	square<<<height-1, width-1>>>(d_out, d_in, d_wid);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//Encoding Image
	lodepng_encode32_file(newFileName, host_imageOutput, width, height);

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}

